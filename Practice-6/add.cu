﻿#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void addKernel(const int* a, const int* b, int* c)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    c[i] = a[i] + b[i];
}